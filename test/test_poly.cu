#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdint>
#include <iostream>

#include "hifive/kernel/polynomial.hpp"

__global__ void gAdd(DeviceContext *dc, const int N, const int block_width,
                     const int block_height, uint64_t *d_c, const uint64_t *d_a,
                     const uint64_t *d_b, bool c_is_shared, bool a_is_shared,
                     bool b_is_shared) {
    uint64_t *d_ci = d_c + blockIdx.x * block_width;
    const uint64_t *d_ai = d_a + blockIdx.x * block_width;
    const uint64_t *d_bi = d_b + blockIdx.x * block_width;
    Add(dc, N, block_width, block_height, d_ci, d_ai, d_bi, false, false,
        false);
}

int test_poly_add(DeviceContext *dc) {
    uint64_t *a, *b, *c;
    uint64_t *d_a, *d_b, *d_c;
    const int N = 1 << 16;
    const int L = 24;
    a = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    b = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    c = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    for (int i = 0; i < N * L; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMalloc(&d_a, N * L * sizeof(uint64_t));
    hipMalloc(&d_b, N * L * sizeof(uint64_t));
    hipMalloc(&d_c, N * L * sizeof(uint64_t));
    hipMemcpy(d_a, a, N * L * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * L * sizeof(uint64_t), hipMemcpyHostToDevice);

    const int block_width = 1 << 8;
    const int block_height = L;
    const int block_size = block_width * block_height * sizeof(uint64_t);
    for (int i = 0; i < 10; i++) {
        auto start = std::chrono::high_resolution_clock::now();
        gAdd<<<N / block_width, block_width, block_size>>>(
            dc, N, block_width, block_height, d_c, d_a, d_b, false, false,
            false);
        CudaCheckError();
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();

        hipMemcpy(c, d_c, N * L * sizeof(uint64_t), hipMemcpyDeviceToHost);
        for (int i = 0; i < N * L; i++) {
            if (c[i] != a[i] + b[i]) {
                std::cout << "Error at index " << i << ": " << c[i]
                          << " != " << a[i] << " + " << b[i] << std::endl;
                return 1;
            }
        }
        auto elapsed =
            std::chrono::duration_cast<std::chrono::microseconds>(end - start);

        std::cout << "Time: " << elapsed.count() << "us" << std::endl;
    }
    return 0;
}