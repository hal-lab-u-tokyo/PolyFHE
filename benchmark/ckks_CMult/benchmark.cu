#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>

#include "phantom-fhe/examples/example.h"
#include "phantom-fhe/include/phantom.h"
#include "phantom-fhe/include/util.cuh"
#include "polyfhe/kernel/device_context.hpp"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

#define EPSINON 0.001

void entry_kernel(Params *params_d, Params *params_h, uint64_t *in0,
                  uint64_t *in1, uint64_t *out0, bool if_benchmark);

inline bool operator==(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return fabs(lhs.x - rhs.x) < EPSINON;
}

inline bool compare_double(const double &lhs, const double &rhs) {
    return fabs(lhs - rhs) < EPSINON;
}

void ConvertPhantomToParams(Params &params, const DModulus *d_modulus) {
    uint64_t *d_tmp;
    hipMalloc(&d_tmp, params.L * sizeof(uint64_t));
    for (int i = 0; i < params.L; i++) {
        hipMemcpy(d_tmp + i, d_modulus[i].data(), sizeof(uint64_t),
                   hipMemcpyDeviceToDevice);
    }
    params.qVec = d_tmp;

    uint64_t *d_modulus_const_ratio;
    hipMalloc(&d_modulus_const_ratio, 2 * params.L * sizeof(uint64_t));
    for (int i = 0; i < params.L; i++) {
        hipMemcpy(d_modulus_const_ratio + 2 * i, d_modulus[i].const_ratio(),
                   2 * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    }
    params.modulus_const_ratio = d_modulus_const_ratio;
}

void example_ckks(PhantomContext &context, const double &scale) {
    // KeyGen
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomCKKSEncoder encoder(context);

    size_t slot_count = encoder.slot_count();
    vector<hipDoubleComplex> input1, input2_const, result;
    size_t msg_size1 = slot_count;
    size_t msg_size2 = slot_count;
    input1.reserve(msg_size1);
    input2_const.reserve(msg_size2);
    double rand_real, rand_imag;
    srand(time(0));
    for (size_t i = 0; i < msg_size1; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input1.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }
    for (size_t i = 0; i < msg_size2; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input2_const.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }

    cout << "Input vector 1: length = " << msg_size1 << endl;
    print_vector(input1, 3, 7);
    cout << "Input vector 2: length = " << msg_size2 << endl;
    print_vector(input2_const, 3, 7);

    PhantomPlaintext x_plain, const_plain;
    encoder.encode(context, input1, scale, x_plain);
    encoder.encode(context, input2_const, scale, const_plain);

    PhantomCiphertext x_sym_cipher;
    secret_key.encrypt_symmetric(context, x_plain, x_sym_cipher);

    uint64_t *in1 = x_sym_cipher.data();
    uint64_t *in2 = const_plain.data();
    uint64_t *res = x_sym_cipher.data();
    uint64_t poly_degree = context.gpu_rns_tables().n();
    auto &context_data = context.get_context_data(x_sym_cipher.chain_index());
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    uint64_t coeff_mod_size = coeff_modulus.size();

    Params params_h;
    params_h.N = poly_degree;
    params_h.L = coeff_mod_size;
    ConvertPhantomToParams(params_h, context.gpu_rns_tables().modulus());
    Params *params_d;
    checkCudaErrors(hipMalloc((void **) &params_d, sizeof(Params)));
    checkCudaErrors(hipMemcpy(params_d, &params_h, sizeof(Params),
                               hipMemcpyHostToDevice));
    entry_kernel(params_d, &params_h, in1, in2, res, true);
}

int main() {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
    PhantomContext context(parms);
    print_parameters(context);
    example_ckks(context, scale);
}