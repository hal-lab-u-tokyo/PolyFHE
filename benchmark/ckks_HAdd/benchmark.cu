#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>

#include "phantom-fhe/examples/example.h"
#include "phantom-fhe/include/phantom.h"
#include "phantom-fhe/include/util.cuh"
#include "polyfhe/kernel/device_context.hpp"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

#define EPSINON 0.001

void entry_kernel(Params *params_d, Params *params_h, uint64_t *in0,
                  uint64_t *in1, uint64_t *out0, bool if_benchmark);

inline bool operator==(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return fabs(lhs.x - rhs.x) < EPSINON;
}

inline bool compare_double(const double &lhs, const double &rhs) {
    return fabs(lhs - rhs) < EPSINON;
}

uint64_t *convert_DModulus_to_uint64_t(const DModulus *d_modulus, int len) {
    uint64_t *d_modulus_new;
    hipMalloc(&d_modulus_new, len * sizeof(uint64_t));
    for (int i = 0; i < len; i++) {
        hipMemcpy(d_modulus_new + i, d_modulus[i].data(), sizeof(uint64_t),
                   hipMemcpyDeviceToDevice);
    }
    return d_modulus_new;
}

void example_ckks(PhantomContext &context, const double &scale) {
    // KeyGen
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomCKKSEncoder encoder(context);

    size_t slot_count = encoder.slot_count();
    vector<hipDoubleComplex> input1, input2, result;
    size_t msg_size1 = slot_count;
    size_t msg_size2 = slot_count;
    input1.reserve(msg_size1);
    input2.reserve(msg_size2);
    double rand_real, rand_imag;
    srand(time(0));
    for (size_t i = 0; i < msg_size1; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input1.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }
    for (size_t i = 0; i < msg_size2; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input2.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }

    PhantomPlaintext x_plain, y_plain;
    encoder.encode(context, input1, scale, x_plain);
    encoder.encode(context, input2, scale, y_plain);

    PhantomCiphertext x_sym_cipher, y_sym_cipher;
    secret_key.encrypt_symmetric(context, x_plain, x_sym_cipher);
    secret_key.encrypt_symmetric(context, y_plain, y_sym_cipher);

    uint64_t *in1 = x_sym_cipher.data();
    uint64_t *in2 = y_sym_cipher.data();
    uint64_t *res = x_sym_cipher.data();
    uint64_t poly_degree = context.gpu_rns_tables().n();
    auto &context_data = context.get_context_data(x_sym_cipher.chain_index());
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    uint64_t coeff_mod_size = coeff_modulus.size();
    uint64_t *modulus = convert_DModulus_to_uint64_t(
        context.gpu_rns_tables().modulus(), coeff_mod_size);
    checkCudaErrors(hipGetLastError());

    Params params_h;
    Params *params_d;
    checkCudaErrors(hipMalloc((void **) &params_d, sizeof(Params)));
    params_h.N = poly_degree;
    params_h.L = coeff_mod_size;
    params_h.qVec = modulus;
    checkCudaErrors(hipMemcpy(params_d, &params_h, sizeof(Params),
                               hipMemcpyHostToDevice));
    entry_kernel(params_d, &params_h, in1, in2, res, true);
}

int main() {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
    PhantomContext context(parms);
    print_parameters(context);
    example_ckks(context, scale);
}