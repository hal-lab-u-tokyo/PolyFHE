#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>

#include "polynomial.h"

// Goal of code generation:
__global__ void fused_add(uint64_t *dst, const uint64_t *a, const uint64_t *b,
                          const int width, const int height, const int N) {
    extern __shared__ uint64_t shared[];
    poly_add(shared, a, b, true, false, false, width, height, N);
    poly_add(dst, shared, b, false, true, false, width, height, N);
}

int test_add() {
    uint64_t *a, *b, *c;
    uint64_t *d_a, *d_b, *d_c;
    const int N = 1 << 16;
    const int L = 24;
    a = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    b = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    c = (uint64_t *) malloc(N * L * sizeof(uint64_t));
    for (int i = 0; i < N * L; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMalloc(&d_a, N * L * sizeof(uint64_t));
    hipMalloc(&d_b, N * L * sizeof(uint64_t));
    hipMalloc(&d_c, N * L * sizeof(uint64_t));
    hipMemcpy(d_a, a, N * L * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * L * sizeof(uint64_t), hipMemcpyHostToDevice);

    const int block_width = 256;
    const int block_height = L;
    const int block_size = block_width * block_height * sizeof(uint64_t);
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < N / block_width; i++) {
        fused_add<<<1, block_width, block_size>>>(
            d_c + i * block_width, d_a + i * block_width, d_b + i * block_width,
            block_width, block_height, N);
    }
    CudaCheckError();
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end -
                                                                       start)
                     .count()
              << "us" << std::endl;
    hipMemcpy(c, d_c, N * L * sizeof(uint64_t), hipMemcpyDeviceToHost);
    for (int i = 0; i < N * L; i++) {
        if (c[i] != a[i] + b[i] * 2) {
            std::cout << "Error at index " << i << ": " << c[i]
                      << " != " << a[i] << " + " << b[i] * 2 << std::endl;
            return 1;
        }
    }
    return 0;
}

int main() {
    int ret;
    ret = test_add();
    if (ret != 0) {
        std::cout << "Test Add failed:  " << ret << std::endl;
        return ret;
    }

    std::cout << "All tests passed" << std::endl;
    return 0;
}