#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>

#include "examples/example.h"
#include "gtest/gtest.h"
#include "phantom.h"
#include "util.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

#define EPSINON 0.001

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

void __checkCudaErrors(hipError_t err, const char *filename, int line);
inline void __checkCudaErrors(hipError_t err, const char *filename, int line) {
    assert(filename);
    if (hipSuccess != err) {
        const char *ename = hipGetErrorName(err);
        printf(
            "CUDA API Error %04d: \"%s\" from file <%s>, "
            "line %i.\n",
            err, ((ename != NULL) ? ename : "Unknown"), filename, line);
        // exit(err);
    }
}

inline bool operator==(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return fabs(lhs.x - rhs.x) < EPSINON;
}

inline bool compare_double(const double &lhs, const double &rhs) {
    return fabs(lhs - rhs) < EPSINON;
}

uint64_t *convert_DModulus_to_uint64_t(const DModulus *d_modulus, int len) {
    uint64_t *d_modulus_new;
    hipMalloc(&d_modulus_new, len * sizeof(uint64_t));
    for (int i = 0; i < len; i++) {
        hipMemcpy(d_modulus_new + i, d_modulus[i].data(), sizeof(uint64_t),
                   hipMemcpyDeviceToDevice);
    }
    return d_modulus_new;
}

__global__ void poly_add(uint64_t *res, uint64_t *in1, uint64_t *in2,
                         uint64_t *modulus, uint64_t degree,
                         uint64_t mod_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < degree * mod_size) {
        uint64_t idx_mod = idx / degree;
        uint64_t idx_poly = idx % degree;
        uint64_t data_idx = idx_mod * degree + idx_poly;
        res[data_idx] = (in1[data_idx] + in2[data_idx]) % modulus[idx_mod];
    }
}

void example_ckks_encode_decode(PhantomContext &context, const double &scale) {
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);

    PhantomCKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();

    // Encode / Decode check
    vector<double> input(slot_count);
    vector<double> result;
    PhantomPlaintext pt;
    for (size_t i = 0; i < slot_count; i++) {
        input[i] = (double) rand() / RAND_MAX;
    }
    encoder.encode(context, input, scale, pt, 1);
    encoder.decode(context, pt, result);

    bool correctness = true;
    for (size_t i = 0; i < slot_count; i++) {
        correctness &= compare_double(result[i], input[i]);
    }
    EXPECT_EQ(correctness, true);
}

void example_ckks_enc_sym(PhantomContext &context, const double &scale) {
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomCKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();

    vector<double> input(slot_count);
    vector<double> result;
    PhantomPlaintext pt;
    for (size_t i = 0; i < slot_count; i++) {
        input[i] = (double) rand() / RAND_MAX;
    }
    encoder.encode(context, input, scale, pt, 1);

    // Symmetric encryption check
    PhantomCiphertext x_symmetric_cipher;
    PhantomPlaintext x_symmetric_plain;
    secret_key.encrypt_symmetric(context, pt, x_symmetric_cipher);
    secret_key.decrypt(context, x_symmetric_cipher, x_symmetric_plain);
    encoder.decode(context, x_symmetric_plain, result);

    bool correctness = true;
    for (size_t i = 0; i < slot_count; i++) {
        correctness &= compare_double(result[i], input[i]);
    }
    EXPECT_EQ(correctness, true);
}

void example_ckks_enc_asymm(PhantomContext &context, const double &scale) {
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomCKKSEncoder encoder(context);
    size_t slot_count = encoder.slot_count();

    vector<double> input(slot_count);
    vector<double> result;
    PhantomPlaintext pt;
    for (size_t i = 0; i < slot_count; i++) {
        input[i] = (double) rand() / RAND_MAX;
    }
    encoder.encode(context, input, scale, pt, 1);

    // Asymmetric encryption check
    PhantomCiphertext x_asymmetric_cipher;
    PhantomPlaintext x_asymmetric_plain;
    public_key.encrypt_asymmetric(context, pt, x_asymmetric_cipher);
    secret_key.decrypt(context, x_asymmetric_cipher, x_asymmetric_plain);
    encoder.decode(context, x_asymmetric_plain, result);

    bool correctness = true;
    for (size_t i = 0; i < slot_count; i++) {
        correctness &= compare_double(result[i], input[i]);
    }
    EXPECT_EQ(correctness, true);
}

void example_ckks_add(PhantomContext &context, const double &scale) {
    // KeyGen
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomCKKSEncoder encoder(context);

    size_t slot_count = encoder.slot_count();
    vector<hipDoubleComplex> input1, input2, result;
    size_t msg_size1 = slot_count;
    size_t msg_size2 = slot_count;
    input1.reserve(msg_size1);
    input2.reserve(msg_size2);
    double rand_real, rand_imag;
    srand(time(0));
    for (size_t i = 0; i < msg_size1; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input1.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }
    for (size_t i = 0; i < msg_size2; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input2.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }

    cout << "Input vector 1: length = " << msg_size1 << endl;
    print_vector(input1, 3, 7);
    cout << "Input vector 2: length = " << msg_size2 << endl;
    print_vector(input2, 3, 7);

    PhantomPlaintext x_plain, y_plain;
    encoder.encode(context, input1, scale, x_plain);
    encoder.encode(context, input2, scale, y_plain);

    PhantomCiphertext x_sym_cipher, y_sym_cipher;
    secret_key.encrypt_symmetric(context, x_plain, x_sym_cipher);
    secret_key.encrypt_symmetric(context, y_plain, y_sym_cipher);

    uint64_t *in1 = x_sym_cipher.data();
    uint64_t *in2 = y_sym_cipher.data();
    uint64_t *res = x_sym_cipher.data();
    uint64_t poly_degree = context.gpu_rns_tables().n();
    auto &context_data = context.get_context_data(x_sym_cipher.chain_index());
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    uint64_t coeff_mod_size = coeff_modulus.size();
    uint64_t *modulus = convert_DModulus_to_uint64_t(
        context.gpu_rns_tables().modulus(), coeff_mod_size);
    checkCudaErrors(hipGetLastError());
    poly_add<<<4096, 512>>>(res, in1, in2, modulus, poly_degree,
                            coeff_mod_size);
    checkCudaErrors(hipDeviceSynchronize());
    in1 = x_sym_cipher.data() + poly_degree * coeff_mod_size;
    in2 = y_sym_cipher.data() + poly_degree * coeff_mod_size;
    res = x_sym_cipher.data() + poly_degree * coeff_mod_size;
    poly_add<<<4096, 512>>>(res, in1, in2, modulus, poly_degree,
                            coeff_mod_size);
    checkCudaErrors(hipDeviceSynchronize());

    PhantomPlaintext x_plus_y_sym_plain;
    secret_key.decrypt(context, x_sym_cipher, x_plus_y_sym_plain);
    encoder.decode(context, x_plus_y_sym_plain, result);

    cout << "Result: " << endl;
    print_vector(result, 3, 7);

    bool correctness = true;
    for (size_t i = 0; i < max(msg_size1, msg_size2); i++) {
        if (i >= msg_size1)
            correctness &= result[i] == input2[i];
        else if (i >= msg_size2)
            correctness &= result[i] == input1[i];
        else
            correctness &= result[i] == hipCadd(input1[i], input2[i]);
    }
    EXPECT_EQ(correctness, true);
}

TEST(TestEncode, logN15_L20_alpha1) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
    PhantomContext context(parms);
    example_ckks_encode_decode(context, scale);
}

TEST(TestEncode, logN15_L18_alpha2) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 60, 60}));
    parms.set_special_modulus_size(2);
    PhantomContext context(parms);
    example_ckks_encode_decode(context, scale);
}

TEST(TestEncode, logN15_L18_alpha3) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 60, 60, 60}));
    parms.set_special_modulus_size(3);
    PhantomContext context(parms);
    example_ckks_encode_decode(context, scale);
}

TEST(TestEncode, logN15_L16_alpha4) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree,
        {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60}));
    parms.set_special_modulus_size(4);
    PhantomContext context(parms);
    example_ckks_encode_decode(context, scale);
}

TEST(TestEncode, logN16_L40_alpha15) {
    srand(time(NULL));
    double scale = pow(2.0, 50);
    size_t poly_modulus_degree = 1 << 16;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree,
        {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60}));
    parms.set_special_modulus_size(15);
    PhantomContext context(parms);
    example_ckks_encode_decode(context, scale);
}

TEST(TestEncrypt, logN15_L20_alpha1) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
    PhantomContext context(parms);
    example_ckks_enc_sym(context, scale);
    example_ckks_enc_asymm(context, scale);
}

TEST(TestEncrypt, logN15_L18_alpha2) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 60, 60}));
    parms.set_special_modulus_size(2);
    PhantomContext context(parms);
    example_ckks_enc_sym(context, scale);
    example_ckks_enc_asymm(context, scale);
}

TEST(TestEncrypt, logN15_L18_alpha3) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 60, 60, 60}));
    parms.set_special_modulus_size(3);
    PhantomContext context(parms);
    example_ckks_enc_sym(context, scale);
    example_ckks_enc_asymm(context, scale);
}

TEST(TestEncrypt, logN15_L16_alpha4) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree,
        {60, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 60, 60, 60, 60}));
    parms.set_special_modulus_size(4);
    PhantomContext context(parms);
    example_ckks_enc_sym(context, scale);
    example_ckks_enc_asymm(context, scale);
}

TEST(TestEncrypt, logN16_L40_alpha15) {
    srand(time(NULL));
    double scale = pow(2.0, 50);
    size_t poly_modulus_degree = 1 << 16;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree,
        {60, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50, 50,
         60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60, 60}));
    parms.set_special_modulus_size(15);
    PhantomContext context(parms);
    example_ckks_enc_sym(context, scale);
    example_ckks_enc_asymm(context, scale);
}

TEST(TestHAdd, logN15_L20_alpha1) {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 40, 60}));
    PhantomContext context(parms);
    print_parameters(context);
    example_ckks_add(context, scale);
}