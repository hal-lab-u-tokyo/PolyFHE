#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <cstddef>
#include <fstream>
#include <iostream>
#include <mutex>
#include <random>
#include <vector>

#include "phantom-fhe/examples/example.h"
#include "phantom-fhe/include/butterfly.cuh"
#include "phantom-fhe/include/phantom.h"
#include "polyfhe/kernel/device_context.hpp"
#include "polyfhe/kernel/ntt-phantom.hpp"
#include "polyfhe/kernel/polynomial.cuh"

using namespace std;
using namespace phantom;
using namespace phantom::arith;
using namespace phantom::util;

#define EPSINON 0.001

void entry_kernel(Params *params_d, Params *params_h, PhantomContext &context,
                  uint64_t *in0, uint64_t *in1, uint64_t *out0, uint64_t *out2,
                  bool if_benchmark);

inline bool operator==(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return fabs(lhs.x - rhs.x) < EPSINON;
}

inline bool compare_double(const double &lhs, const double &rhs) {
    return fabs(lhs - rhs) < EPSINON;
}

/*
__global__ static void BConvGlobal(Params *params, uint64_t *dst,
                                   const uint64_t *in,
                                   const uint64_t *qiHat_mod_pj,
                                   const DModulus *ibase, uint64_t ibase_size,
                                   const DModulus *obase, uint64_t obase_size,
                                   size_t startPartIdx, size_t size_PartQl) {
    extern __shared__ uint64_t shared[];
    // TODO: malloc ibase_size
    uint64_t reg_ibase[8];

    for (size_t i = threadIdx.x; i < obase_size * ibase_size; i += blockDim.x) {
        shared[i] = qiHat_mod_pj[i];
    }
    __syncthreads();

    constexpr const int unroll_number = 2;
    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
         tid < (params->N * obase_size + unroll_number - 1) / unroll_number;
         tid += blockDim.x * gridDim.x) {
        const size_t n_idx = unroll_number * (tid / obase_size);
        const size_t l_idx = tid % obase_size;

        // Load
        for (int i = 0; i < ibase_size; i++) {
            reg_ibase[2 * i] = *(in + params->N * i + n_idx);
            reg_ibase[2 * i + 1] = *(in + params->N * i + n_idx + 1);
        }

        uint64_t res1, res2;

        BConvOp(params, &res1, &res2, reg_ibase, shared, n_idx, l_idx, ibase,
                ibase_size, obase, obase_size, startPartIdx, size_PartQl);

        // Leap over the overlapped region.
        const size_t l_out_idx =
            l_idx + ((l_idx >= startPartIdx) ? size_PartQl : 0);
        phantom::arith::st_two_uint64(dst + l_out_idx * params->N + n_idx, res1,
                                      res2);
    }
}
*/

void ConvertPhantomToParams(Params &params, const PhantomContext &context) {
    const DModulus *d_modulus = context.gpu_rns_tables().modulus();
    const DNTTTable &ntt_tables = context.gpu_rns_tables();
    uint64_t *d_tmp;
    hipMalloc(&d_tmp, params.L * sizeof(uint64_t));
    for (int i = 0; i < params.L; i++) {
        hipMemcpy(d_tmp + i, d_modulus[i].data(), sizeof(uint64_t),
                   hipMemcpyDeviceToDevice);
    }
    params.qVec = d_tmp;

    uint64_t *d_modulus_const_ratio;
    hipMalloc(&d_modulus_const_ratio, 2 * params.L * sizeof(uint64_t));
    for (int i = 0; i < params.L; i++) {
        hipMemcpy(d_modulus_const_ratio + 2 * i, d_modulus[i].const_ratio(),
                   2 * sizeof(uint64_t), hipMemcpyDeviceToDevice);
    }
    params.modulus_const_ratio = d_modulus_const_ratio;

    // NTT tables
    params.itwiddle = ntt_tables.itwiddle();
    params.itwiddle_shoup = ntt_tables.itwiddle_shoup();
    params.n_inv = ntt_tables.n_inv_mod_q();
    params.n_inv_shoup = ntt_tables.n_inv_mod_q_shoup();

    // DRNSTool
    for (int i = 0; i < params.L + 1; i++) {
        auto &context_data = context.get_context_data(i);
        phantom::DRNSTool &rns_tool = context_data.gpu_rns_tool();
        params.rns_tools.push_back(&rns_tool);
    }
}

void example_ckks(PhantomContext &context, const double &scale) {
    // KeyGen
    PhantomSecretKey secret_key(context);
    PhantomPublicKey public_key = secret_key.gen_publickey(context);
    PhantomRelinKey relin_keys = secret_key.gen_relinkey(context);
    PhantomCKKSEncoder encoder(context);
    const auto &s = hipStreamPerThread;

    size_t slot_count = encoder.slot_count();
    vector<hipDoubleComplex> input1, input2, result;
    size_t msg_size1 = slot_count;
    size_t msg_size2 = slot_count;
    input1.reserve(msg_size1);
    input2.reserve(msg_size2);
    double rand_real, rand_imag;
    srand(time(0));
    for (size_t i = 0; i < msg_size1; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input1.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }
    for (size_t i = 0; i < msg_size2; i++) {
        rand_real = (double) rand() / RAND_MAX;
        rand_imag = (double) rand() / RAND_MAX;
        input2.push_back(make_hipDoubleComplex(rand_real, rand_imag));
    }

    cout << "Input vector 1: length = " << msg_size1 << endl;
    print_vector(input1, 3, 7);
    cout << "Input vector 2: length = " << msg_size2 << endl;
    print_vector(input2, 3, 7);

    PhantomPlaintext x_plain, y_plain;
    encoder.encode(context, input1, scale, x_plain);
    encoder.encode(context, input2, scale, y_plain);
    std::cout << "x_plain.chain_index(): " << x_plain.chain_index()
              << std::endl;

    PhantomCiphertext x_cipher, y_cipher;
    secret_key.encrypt_symmetric(context, x_plain, x_cipher);
    secret_key.encrypt_symmetric(context, y_plain, y_cipher);
    std::cout << "x_plain.chain_index(): " << x_plain.chain_index()
              << std::endl;
    std::cout << "x_cipher.chain_index(): " << x_cipher.chain_index()
              << std::endl;

    // PolyFHE's HMult
    PhantomCiphertext xy_cipher_polyfhe = x_cipher;
    uint64_t poly_degree = context.gpu_rns_tables().n();
    auto &context_data = context.get_context_data(x_cipher.chain_index());
    auto &parms = context_data.parms();
    auto &coeff_modulus = parms.coeff_modulus();
    uint64_t coeff_mod_size = coeff_modulus.size();

    std::cout << "coeff_mod_size: " << coeff_mod_size << std::endl;

    Params params_h(std::log2(poly_degree), coeff_mod_size, 9);
    ConvertPhantomToParams(params_h, context);
    Params *params_d;
    checkCudaErrors(hipMalloc((void **) &params_d, sizeof(Params)));
    checkCudaErrors(hipMemcpy(params_d, &params_h, sizeof(Params),
                               hipMemcpyHostToDevice));

    uint64_t *in1 = x_cipher.data();
    uint64_t *in2 = y_cipher.data();
    xy_cipher_polyfhe.resize(3, coeff_mod_size, poly_degree, s);
    uint64_t *res = xy_cipher_polyfhe.data();

    const int beta = std::ceil((params_h.L + 1) / params_h.alpha);
    const int sizeQP = coeff_mod_size + params_h.alpha;
    const int sizeQPNBeta = poly_degree * sizeQP * beta;
    std::cout << "beta: " << beta << std::endl;
    uint64_t *res_modup_polyfhe, *res_modup_polyfhe2, *res_modup_phantom;
    checkCudaErrors(hipMalloc((void **) &res_modup_polyfhe,
                               sizeQPNBeta * sizeof(uint64_t)));
    checkCudaErrors(hipMalloc((void **) &res_modup_polyfhe2,
                               sizeQPNBeta * sizeof(uint64_t)));
    checkCudaErrors(hipMalloc((void **) &res_modup_phantom,
                               sizeQPNBeta * sizeof(uint64_t)));

    // PolyFHE's HMult
    entry_kernel(params_d, &params_h, context, in1, in2, res, res_modup_polyfhe,
                 true);
    checkCudaErrors(hipDeviceSynchronize());
    /*
    phantom::DRNSTool *drns_tool = params_h.rns_tools[1];
    for (size_t beta_idx = 0; beta_idx < beta; beta_idx++) {
        const size_t startPartIdx = params_h.alpha * beta_idx;
        const size_t size_PartQl =
            (beta_idx == beta - 1) ? (params_h.L - params_h.alpha * (beta - 1))
                                   : params_h.alpha;

        const uint64_t *in_modup_i =
            res_modup_polyfhe + poly_degree * startPartIdx;
        uint64_t *out_modup_i =
            res_modup_polyfhe2 + poly_degree * beta_idx * sizeQP;

        auto &bconv_pre =
            drns_tool->v_base_part_Ql_to_compl_part_QlP_conv()[beta_idx];
        auto &ibase = bconv_pre.ibase();
        auto &obase = bconv_pre.obase();
        const auto qiHat_mod_pj = bconv_pre.QHatModp();

        uint64_t gridDimGlb;
        constexpr int unroll_factor = 2;
        gridDimGlb = params_h.N * obase.size() / blockDimGlb.x / unroll_factor;
        std::cout << "beta_idx: " << beta_idx << std::endl;
        BConvGlobal<<<gridDimGlb, blockDimGlb,
                      sizeof(uint64_t) * obase.size() * ibase.size()>>>(
            params_d, out_modup_i, in_modup_i, qiHat_mod_pj, ibase.base(),
            ibase.size(), obase.base(), obase.size(), startPartIdx,
            size_PartQl);
        checkCudaErrors(hipDeviceSynchronize());
    }
    */
    checkCudaErrors(hipDeviceSynchronize());

    // Phantom's HMult
    PhantomCiphertext xy_cipher = multiply(context, x_cipher, y_cipher);
    relinearize_inplace_debug(context, xy_cipher, relin_keys,
                              res_modup_phantom);
    checkCudaErrors(hipDeviceSynchronize());
    // rescale_to_next_inplace(context, xy_cipher);
    std::cout << "xy_cipher.chain_index(): " << xy_cipher.chain_index()
              << std::endl;

    /*
     */
    // Check if PolyFHE's HMult and Phantom's HMult are the same
    uint64_t *h_res_polyfhe =
        (uint64_t *) malloc(poly_degree * coeff_mod_size * sizeof(uint64_t));
    uint64_t *h_res_phantom =
        (uint64_t *) malloc(poly_degree * coeff_mod_size * sizeof(uint64_t));

    bool correctness = true;
    for (int idx = 0; idx < xy_cipher.size(); idx++) {
        std::cout << "idx: " << idx << std::endl;
        correctness = true;
        uint64_t *d_res_polyfhe =
            xy_cipher_polyfhe.data() + idx * poly_degree * coeff_mod_size;
        uint64_t *d_res_phantom =
            xy_cipher.data() + idx * poly_degree * coeff_mod_size;
        checkCudaErrors(
            hipMemcpy(h_res_polyfhe, d_res_polyfhe,
                       poly_degree * coeff_mod_size * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
        checkCudaErrors(
            hipMemcpy(h_res_phantom, d_res_phantom,
                       poly_degree * coeff_mod_size * sizeof(uint64_t),
                       hipMemcpyDeviceToHost));
        for (int i = 0; i < poly_degree * coeff_mod_size; i++) {
            if (h_res_polyfhe[i] != h_res_phantom[i]) {
                correctness = false;
                cout << "  PolyFHE != Phantom at index " << i << endl;
                cout << "   PolyFHE: " << h_res_polyfhe[i] << endl;
                cout << "   Phantom: " << h_res_phantom[i] << endl;
                break;
            }
        }
        if (correctness) {
            cout << "  OK" << endl;
        } else {
            cout << "  Fail" << endl;
        }
    }

    // Check t_modup_ptr
    uint64_t *h_modup_polyfhe =
        (uint64_t *) malloc(sizeQPNBeta * sizeof(uint64_t));
    uint64_t *h_modup_phantom =
        (uint64_t *) malloc(sizeQPNBeta * sizeof(uint64_t));
    checkCudaErrors(hipMemcpy(h_modup_polyfhe, res_modup_polyfhe,
                               sizeQPNBeta * sizeof(uint64_t),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_modup_phantom, res_modup_phantom,
                               sizeQPNBeta * sizeof(uint64_t),
                               hipMemcpyDeviceToHost));
    std::cout << "Modup result" << std::endl;
    correctness = true;
    for (int beta_idx = 0; beta_idx < beta; beta_idx++) {
        for (int j = 0; j < poly_degree * params_h.KL; j++) {
            int i = beta_idx * poly_degree * params_h.KL + j;
            if (h_modup_polyfhe[i] != h_modup_phantom[i]) {
                cout << "  PolyFHE != Phantom at index[" << beta_idx << "]["
                     << j << "]" << endl;
                cout << "   PolyFHE: " << h_modup_polyfhe[i] << endl;
                cout << "   Phantom: " << h_modup_phantom[i] << endl;
                correctness = false;
                break;
            }
        }
    }
    if (correctness) {
        cout << "  OK" << endl;
    } else {
        cout << "  Fail" << endl;
    }

    std::vector<double> elapsed_list;
    for (int iter = 0; iter < 7; iter++) {
        auto start = std::chrono::high_resolution_clock::now();
        PhantomCiphertext xy_cipher = multiply(context, x_cipher, y_cipher);
        relinearize_inplace_debug(context, xy_cipher, relin_keys,
                                  res_modup_phantom);
        checkCudaErrors(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double elapsed =
            std::chrono::duration_cast<std::chrono::microseconds>(end - start)
                .count();
        if (iter != 0) {
            elapsed_list.push_back(elapsed);
        }
    }
    double avg_time =
        std::accumulate(elapsed_list.begin(), elapsed_list.end(), 0.0) /
        elapsed_list.size();
    std::cout << "Average elapsed time (Phantom): " << avg_time << " us"
              << std::endl;

    /*
    PhantomPlaintext xy_plain;
    secret_key.decrypt(context, xy_cipher, xy_plain);
    encoder.decode(context, xy_plain, result);

    cout << "Result: " << endl;
    print_vector(result, 3, 7);

    bool correctness = true;
    for (size_t i = 0; i < max(msg_size1, msg_size2); i++) {
        if (i >= msg_size1)
            correctness &= result[i] == input2[i];
        else if (i >= msg_size2)
            correctness &= result[i] == input1[i];
        else
            correctness &= result[i] == hipCmul(input1[i], input2[i]);
    }
    if (correctness) {
        cout << "Correctness check passed!" << endl;
    } else {
        cout << "Correctness check failed!" << endl;
    }
     */
}

int main() {
    srand(time(NULL));
    double scale = pow(2.0, 40);
    size_t poly_modulus_degree = 1 << 15;
    EncryptionParameters parms(scheme_type::ckks);
    parms.set_poly_modulus_degree(poly_modulus_degree);
    parms.set_coeff_modulus(CoeffModulus::Create(
        poly_modulus_degree, {60, 40, 40, 40, 40, 40, 40, 40, 40, 40,
                              40, 40, 40, 40, 40, 40, 40, 40, 60, 60}));
    parms.set_special_modulus_size(2);
    PhantomContext context(parms);
    print_parameters(context);
    example_ckks(context, scale);
}