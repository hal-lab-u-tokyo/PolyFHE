#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include <chrono>

__device__ void Scale2(uint64_t *a, int n, int l) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n * l;
         idx += blockDim.x * gridDim.x) {
        a[idx] = a[idx] * 2;
    }
}

__device__ void Scale2Limb(uint64_t *a_i, int n) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n;
         idx += blockDim.x * gridDim.x) {
        a_i[idx] = a_i[idx] * 2;
    }
}

__global__ void Scale2Twice(uint64_t *a, int n, int l) {
    Scale2(a, n, l);
    Scale2(a, n, l);
}

__global__ void Scale2TwiceLimbByLimb(uint64_t *a, int n, int l) {
    for (int i = 0; i < l; i++) {
        Scale2Limb(a + i * n, n);
        Scale2Limb(a + i * n, n);
    }
}

enum class ParamSize {
    Small,
    Medium,
    Large,
};

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <if_opt>\n", argv[0]);
        return -1;
    }
    bool if_opt = atoi(argv[1]);
    printf("if_opt: %d\n", if_opt);

    int N, L;
    ParamSize param_size = ParamSize::Small;
    switch (param_size) {
    case ParamSize::Small:
        N = 1 << 15;
        L = 10;
        break;
    case ParamSize::Medium:
        N = 1 << 16;
        L = 20;
        break;
    case ParamSize::Large:
        N = 1 << 17;
        L = 50;
        break;
    default:
        printf("Invalid parameter size\n");
        return -1;
    }

    uint64_t *a, *d_a;
    size_t size = N * L * sizeof(uint64_t);
    a = (uint64_t *) malloc(size);
    hipMalloc((void **) &d_a, size);
    for (int i = 0; i < N * L; i++) {
        a[i] = i;
    }
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    if (if_opt) {
        Scale2TwiceLimbByLimb<<<4096, 128>>>(d_a, N, L);
        hipDeviceSynchronize();
    } else {
        Scale2Twice<<<4096, 128>>>(d_a, N, L);
        hipDeviceSynchronize();
    }

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < N * L; i++) {
        if (a[i] != i * 4) {
            printf("Error at index %d: expected %d, got %lu\n", i, i * 2, a[i]);
            break;
        }
    }
    printf("OK\n");

    double sum = 0;
    for (int iter = 0; iter < 10; iter++) {
        auto start = std::chrono::high_resolution_clock::now();
        if (if_opt) {
            Scale2TwiceLimbByLimb<<<4096, 128>>>(d_a, N, L);
            hipDeviceSynchronize();
        } else {
            Scale2Twice<<<4096, 128>>>(d_a, N, L);
            hipDeviceSynchronize();
        }
        auto end = std::chrono::high_resolution_clock::now();
        auto us =
            std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        sum += us.count();
        printf("Iteration %d: %ld us\n", iter, us.count());
    }
    printf("Average time: %lf us\n", sum / 10);
}