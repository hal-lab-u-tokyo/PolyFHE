#include "device_context.hpp"
#include "hifive/core/param.hpp"
#include "hifive/kernel/FullRNS-HEAAN/src/Context.h"
#include "hifive/kernel/FullRNS-HEAAN/src/EvaluatorUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Numb.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Scheme.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SchemeAlgo.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SecretKey.h"
#include "hifive/kernel/FullRNS-HEAAN/src/StringUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/TimeUtils.h"
#include "hifive/kernel/device_context.hpp"

void FHEContext::Init(const int logN, const int L) {
    LOG_INFO("Initializing DeviceContext\n");
    const uint64_t logp = 55;
    HEAANContext heaan_context(logN, logp, L, L + 1);
    SecretKey secretKey(heaan_context);
    Scheme scheme(secretKey, heaan_context);
    Key key = scheme.keyMap.at(MULTIPLICATION);

    d_context_in_cpu = std::make_shared<DeviceContext>(heaan_context);
    h_context = std::make_shared<HEAANContext>(heaan_context);

    // Copy context_d to device
    checkCudaErrors(
        hipMalloc((void **) &d_context_in_gpu, sizeof(DeviceContext)));
    checkCudaErrors(hipMemcpy(d_context_in_gpu, d_context_in_cpu.get(),
                               sizeof(DeviceContext), hipMemcpyHostToDevice));
}

FHEContext::FHEContext() { Init(hifive::logN, hifive::L); }

FHEContext::FHEContext(const int logN, const int L) { Init(logN, L); }

uint64_t NTTSampleSize(const uint64_t logN) {
    if (logN == 12) {
        return 1 << 6;
    } else if (logN == 13) {
        return 1 << 7;
    } else if (logN == 14) {
        return 1 << 7;
    } else if (logN == 15) {
        return 1 << 8;
    } else if (logN == 16) {
        return 1 << 8;
    } else if (logN == 17) {
        return 1 << 9;
    } else {
        LOG_ERROR("Invalid logN: %ld\n", logN);
        return 0;
    }
}

uint64_t Inverse(const uint64_t op, const uint64_t prime) {
    uint64_t tmp = op > prime ? (op % prime) : op;
    return powMod(tmp, prime - 2, prime);
}

DeviceContext::DeviceContext(HEAANContext &context) {
    logN = context.logN;
    logNh = context.logNh;
    L = context.L;
    K = context.K;
    N = context.N;
    N1 = NTTSampleSize(logN);
    N2 = N / N1;
    M = context.M;
    Nh = context.Nh;
    logp = context.logp;
    p = context.p;
    h = context.h;
    sigma = context.sigma;

    // qVec, qrVec, qTwok, qkVec, qdVec
    // pVec, prVec, pTwok, pkVec, pdVec
    const long size_L = L * sizeof(uint64_t);
    const long size_K = K * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qrVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qTwok, size_L));
    checkCudaErrors(hipMalloc((void **) &qkVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qdVec, size_L));
    checkCudaErrors(hipMalloc((void **) &pVec, size_K));
    checkCudaErrors(hipMalloc((void **) &prVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pTwok, size_K));
    checkCudaErrors(hipMalloc((void **) &pkVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pdVec, size_K));
    checkCudaErrors(
        hipMemcpy(qVec, context.qVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qrVec, context.qrVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qTwok, context.qTwok, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qkVec, context.qkVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qdVec, context.qdVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pVec, context.pVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(prVec, context.prVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pTwok, context.pTwok, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pkVec, context.pkVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pdVec, context.pdVec, size_K, hipMemcpyHostToDevice));

    // qRoots, qRootsInv, qRootPows, qRootPowsInv
    // pRoots, pRootsInv, pRootPows, pRootPowsInv
    const uint64_t size_N = N * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qRoots, size_L));
    checkCudaErrors(hipMalloc((void **) &qRootsInv, size_L));
    checkCudaErrors(hipMalloc((void **) &pRoots, size_K));
    checkCudaErrors(hipMalloc((void **) &pRootsInv, size_K));
    checkCudaErrors(
        hipMemcpy(qRoots, context.qRoots, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(qRootsInv, context.qRootsInv, size_L,
                               hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pRoots, context.pRoots, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pRootsInv, context.pRootsInv, size_L,
                               hipMemcpyHostToDevice));

    uint64_t **tmp = new uint64_t *[L];
    uint64_t **tmp2 = new uint64_t *[L];
    uint64_t *tmpN = new uint64_t[N];
    uint64_t *tmpN2 = new uint64_t[N];

    // qRootPows
    for (int i = 0; i < L; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.qRootPows[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPows, size_L));
    checkCudaErrors(hipMemcpy(qRootPows, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsShoup
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < N; j++) {
            Shoup(tmpN[j], context.qRootPows[i][j], context.qVec[i]);
        }
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(
            hipMemcpy(tmp[i], tmpN, size_N, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsShoup, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsShoup, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsInv
    for (int i = 0; i < L; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.qRootPowsInv[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsInv, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsInv, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsInvShoup
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < N; j++) {
            Shoup(tmpN2[j], context.qRootPowsInv[i][j], context.qVec[i]);
        }
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(
            hipMemcpy(tmp[i], tmpN, size_N, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsInvShoup, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsInvShoup, tmp, size_L, hipMemcpyHostToDevice));

    tmp = new uint64_t *[K];
    // pRootPows
    for (int i = 0; i < K; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.pRootPows[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &pRootPows, size_K));
    checkCudaErrors(hipMemcpy(pRootPows, tmp, size_K, hipMemcpyHostToDevice));

    // pRootPowsInv
    for (int i = 0; i < K; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.pRootPowsInv[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &pRootPowsInv, size_K));
    checkCudaErrors(
        hipMemcpy(pRootPowsInv, tmp, size_K, hipMemcpyHostToDevice));
    LOG_INFO("DeviceContext initialized\n");
}