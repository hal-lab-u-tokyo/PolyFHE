#include "hifive/kernel/FullRNS-HEAAN/src/Context.h"
#include "hifive/kernel/FullRNS-HEAAN/src/EvaluatorUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Numb.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Scheme.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SchemeAlgo.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SecretKey.h"
#include "hifive/kernel/FullRNS-HEAAN/src/StringUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/TimeUtils.h"
#include "hifive/kernel/device_context.hpp"

DeviceContext::DeviceContext() {
    LOG_INFO("Initializing DeviceContext\n");
    const int logN = 15;
    // const int N = (1 << logN);
    const int L = 44;
    const uint64_t logp = 55;
    // const uint64_t logSlots = 3;
    // const uint64_t slots = (1 << logSlots);
    Context context(logN, logp, L, L + 1);
    SecretKey secretKey(context);
    Scheme scheme(secretKey, context);
    Key key = scheme.keyMap.at(MULTIPLICATION);

    set_params(context);
}

void DeviceContext::set_params(Context &context) {
    logN = context.logN;
    logNh = context.logNh;
    L = context.L;
    K = context.K;
    N = context.N;
    M = context.M;
    Nh = context.Nh;
    logp = context.logp;
    p = context.p;
    h = context.h;
    sigma = context.sigma;

    // qVec, qrVec, qTwok, qkVec, qdVec
    // pVec, prVec, pTwok, pkVec, pdVec
    const long size_L = L * sizeof(uint64_t);
    const long size_K = K * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qrVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qTwok, size_L));
    checkCudaErrors(hipMalloc((void **) &qkVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qdVec, size_L));
    checkCudaErrors(hipMalloc((void **) &pVec, size_K));
    checkCudaErrors(hipMalloc((void **) &prVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pTwok, size_K));
    checkCudaErrors(hipMalloc((void **) &pkVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pdVec, size_K));
    checkCudaErrors(
        hipMemcpy(qVec, context.qVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qrVec, context.qrVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qTwok, context.qTwok, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qkVec, context.qkVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qdVec, context.qdVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pVec, context.pVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(prVec, context.prVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pTwok, context.pTwok, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pkVec, context.pkVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pdVec, context.pdVec, size_K, hipMemcpyHostToDevice));
}