#include "device_context.hpp"
#include "hifive/core/param.hpp"
#include "hifive/kernel/FullRNS-HEAAN/src/Context.h"
#include "hifive/kernel/FullRNS-HEAAN/src/EvaluatorUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Numb.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Scheme.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SchemeAlgo.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SecretKey.h"
#include "hifive/kernel/FullRNS-HEAAN/src/StringUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/TimeUtils.h"
#include "hifive/kernel/device_context.hpp"

FHEContext::FHEContext() {
    LOG_INFO("Initializing DeviceContext\n");
    const uint64_t logp = 55;
    // const uint64_t logSlots = 3;
    // const uint64_t slots = (1 << logSlots);
    HEAANContext heaan_context(hifive::logN, logp, hifive::L, hifive::L + 1);
    SecretKey secretKey(heaan_context);
    Scheme scheme(secretKey, heaan_context);
    Key key = scheme.keyMap.at(MULTIPLICATION);

    d_context_in_cpu = std::make_shared<DeviceContext>(heaan_context);
    h_context = std::make_shared<HEAANContext>(heaan_context);

    // Copy context_d to device
    checkCudaErrors(
        hipMalloc((void **) &d_context_in_gpu, sizeof(DeviceContext)));
    checkCudaErrors(hipMemcpy(d_context_in_gpu, d_context_in_cpu.get(),
                               sizeof(DeviceContext), hipMemcpyHostToDevice));
}

uint64_t Inverse(const uint64_t op, const uint64_t prime) {
    uint64_t tmp = op > prime ? (op % prime) : op;
    return powMod(tmp, prime - 2, prime);
}

DeviceContext::DeviceContext(HEAANContext &context) {
    logN = context.logN;
    logNh = context.logNh;
    L = context.L;
    K = context.K;
    N = context.N;
    N1 = hifive::N1;
    N2 = hifive::N2;
    M = context.M;
    Nh = context.Nh;
    logp = context.logp;
    p = context.p;
    h = context.h;
    sigma = context.sigma;

    // qVec, qrVec, qTwok, qkVec, qdVec
    // pVec, prVec, pTwok, pkVec, pdVec
    const long size_L = L * sizeof(uint64_t);
    const long size_K = K * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qrVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qTwok, size_L));
    checkCudaErrors(hipMalloc((void **) &qkVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qdVec, size_L));
    checkCudaErrors(hipMalloc((void **) &pVec, size_K));
    checkCudaErrors(hipMalloc((void **) &prVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pTwok, size_K));
    checkCudaErrors(hipMalloc((void **) &pkVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pdVec, size_K));
    checkCudaErrors(
        hipMemcpy(qVec, context.qVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qrVec, context.qrVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qTwok, context.qTwok, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qkVec, context.qkVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qdVec, context.qdVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pVec, context.pVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(prVec, context.prVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pTwok, context.pTwok, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pkVec, context.pkVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pdVec, context.pdVec, size_K, hipMemcpyHostToDevice));

    // qRoots, qRootsInv, qRootPows, qRootPowsInv
    // pRoots, pRootsInv, pRootPows, pRootPowsInv
    const uint64_t size_N = N * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qRoots, size_L));
    checkCudaErrors(hipMalloc((void **) &qRootsInv, size_L));
    checkCudaErrors(hipMalloc((void **) &pRoots, size_K));
    checkCudaErrors(hipMalloc((void **) &pRootsInv, size_K));
    checkCudaErrors(
        hipMemcpy(qRoots, context.qRoots, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(qRootsInv, context.qRootsInv, size_L,
                               hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pRoots, context.pRoots, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pRootsInv, context.pRootsInv, size_L,
                               hipMemcpyHostToDevice));

    uint64_t **tmp = new uint64_t *[L];
    uint64_t **tmp2 = new uint64_t *[L];
    uint64_t *tmpN = new uint64_t[N];
    uint64_t *tmpN2 = new uint64_t[N];

    // qRootPows
    for (int i = 0; i < L; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.qRootPows[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPows, size_L));
    checkCudaErrors(hipMemcpy(qRootPows, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsShoup
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < N; j++) {
            Shoup(tmpN[j], context.qRootPows[i][j], context.qVec[i]);
        }
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(
            hipMemcpy(tmp[i], tmpN, size_N, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsShoup, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsShoup, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsInv
    for (int i = 0; i < L; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.qRootPowsInv[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsInv, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsInv, tmp, size_L, hipMemcpyHostToDevice));

    // qRootPowsInvShoup
    for (int i = 0; i < L; i++) {
        for (int j = 0; j < N; j++) {
            Shoup(tmpN2[j], context.qRootPowsInv[i][j], context.qVec[i]);
        }
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(
            hipMemcpy(tmp[i], tmpN, size_N, hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &qRootPowsInvShoup, size_L));
    checkCudaErrors(
        hipMemcpy(qRootPowsInvShoup, tmp, size_L, hipMemcpyHostToDevice));

    tmp = new uint64_t *[K];
    // pRootPows
    for (int i = 0; i < K; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.pRootPows[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &pRootPows, size_K));
    checkCudaErrors(hipMemcpy(pRootPows, tmp, size_K, hipMemcpyHostToDevice));

    // pRootPowsInv
    for (int i = 0; i < K; i++) {
        checkCudaErrors(hipMalloc((void **) &tmp[i], size_N));
        checkCudaErrors(hipMemcpy(tmp[i], context.pRootPowsInv[i], size_N,
                                   hipMemcpyHostToDevice));
    }
    checkCudaErrors(hipMalloc((void **) &pRootPowsInv, size_K));
    checkCudaErrors(
        hipMemcpy(pRootPowsInv, tmp, size_K, hipMemcpyHostToDevice));
    LOG_INFO("DeviceContext initialized\n");
}