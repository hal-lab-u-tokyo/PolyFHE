#include "device_context.hpp"
#include "hifive/kernel/FullRNS-HEAAN/src/Context.h"
#include "hifive/kernel/FullRNS-HEAAN/src/EvaluatorUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Numb.h"
#include "hifive/kernel/FullRNS-HEAAN/src/Scheme.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SchemeAlgo.h"
#include "hifive/kernel/FullRNS-HEAAN/src/SecretKey.h"
#include "hifive/kernel/FullRNS-HEAAN/src/StringUtils.h"
#include "hifive/kernel/FullRNS-HEAAN/src/TimeUtils.h"
#include "hifive/kernel/device_context.hpp"

FHEContext::FHEContext() {
    LOG_INFO("Initializing DeviceContext\n");
    const int logN = 15;
    // const int N = (1 << logN);
    const int L = 44;
    const uint64_t logp = 55;
    // const uint64_t logSlots = 3;
    // const uint64_t slots = (1 << logSlots);
    HEAANContext heaan_context(logN, logp, L, L + 1);
    SecretKey secretKey(heaan_context);
    Scheme scheme(secretKey, heaan_context);
    Key key = scheme.keyMap.at(MULTIPLICATION);

    d_context_in_cpu = std::make_shared<DeviceContext>(heaan_context);
    h_context = std::make_shared<HEAANContext>(heaan_context);

    // Copy context_d to device
    checkCudaErrors(
        hipMalloc((void **) &d_context_in_gpu, sizeof(DeviceContext)));
    checkCudaErrors(hipMemcpy(d_context_in_gpu, d_context_in_cpu.get(),
                               sizeof(DeviceContext), hipMemcpyHostToDevice));
}

DeviceContext::DeviceContext(HEAANContext &context) {
    logN = context.logN;
    logNh = context.logNh;
    L = context.L;
    K = context.K;
    N = context.N;
    M = context.M;
    Nh = context.Nh;
    logp = context.logp;
    p = context.p;
    h = context.h;
    sigma = context.sigma;

    // qVec, qrVec, qTwok, qkVec, qdVec
    // pVec, prVec, pTwok, pkVec, pdVec
    const long size_L = L * sizeof(uint64_t);
    const long size_K = K * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qrVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qTwok, size_L));
    checkCudaErrors(hipMalloc((void **) &qkVec, size_L));
    checkCudaErrors(hipMalloc((void **) &qdVec, size_L));
    checkCudaErrors(hipMalloc((void **) &pVec, size_K));
    checkCudaErrors(hipMalloc((void **) &prVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pTwok, size_K));
    checkCudaErrors(hipMalloc((void **) &pkVec, size_K));
    checkCudaErrors(hipMalloc((void **) &pdVec, size_K));
    checkCudaErrors(
        hipMemcpy(qVec, context.qVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qrVec, context.qrVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qTwok, context.qTwok, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qkVec, context.qkVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(qdVec, context.qdVec, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pVec, context.pVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(prVec, context.prVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pTwok, context.pTwok, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pkVec, context.pkVec, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pdVec, context.pdVec, size_K, hipMemcpyHostToDevice));

    // qRoots, qRootsInv, qRootPows, qRootPowsInv
    // pRoots, pRootsInv, pRootPows, pRootPowsInv
    const uint64_t size_NL = N * L * sizeof(uint64_t);
    const uint64_t size_NK = N * K * sizeof(uint64_t);
    checkCudaErrors(hipMalloc((void **) &qRoots, size_L));
    checkCudaErrors(hipMalloc((void **) &qRootsInv, size_L));
    checkCudaErrors(hipMalloc((void **) &qRootPows, size_NL));
    checkCudaErrors(hipMalloc((void **) &qRootPowsInv, size_NL));
    checkCudaErrors(hipMalloc((void **) &pRoots, size_K));
    checkCudaErrors(hipMalloc((void **) &pRootsInv, size_K));
    checkCudaErrors(hipMalloc((void **) &pRootPows, size_NK));
    checkCudaErrors(hipMalloc((void **) &pRootPowsInv, size_NK));
    checkCudaErrors(
        hipMemcpy(qRoots, context.qRoots, size_L, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(qRootsInv, context.qRootsInv, size_L,
                               hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(pRoots, context.pRoots, size_K, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(pRootsInv, context.pRootsInv, size_L,
                               hipMemcpyHostToDevice));
    for (int i = 0; i < L; i++) {
        checkCudaErrors(hipMemcpy(qRootPows + i * N, context.qRootPows[i],
                                   N * sizeof(uint64_t),
                                   hipMemcpyHostToDevice));
        checkCudaErrors(
            hipMemcpy(qRootPowsInv + i * N, context.qRootPowsInv[i],
                       N * sizeof(uint64_t), hipMemcpyHostToDevice));
    }
    for (int i = 0; i < K; i++) {
        checkCudaErrors(hipMemcpy(pRootPows + i * N, context.pRootPows[i],
                                   N * sizeof(uint64_t),
                                   hipMemcpyHostToDevice));
        checkCudaErrors(
            hipMemcpy(pRootPowsInv + i * N, context.pRootPowsInv[i],
                       N * sizeof(uint64_t), hipMemcpyHostToDevice));
    }
}