#include "hip/hip_runtime.h"
#include <csignal>

#include "hifive/kernel/polynomial.hpp"

struct uint128_t {
    uint64_t hi = 0;
    uint64_t lo = 0;
    __device__ uint128_t &operator+=(const uint128_t &op);
    __device__ uint128_t &operator-=(const uint128_t &op);
};

__inline__ __device__ uint128_t &uint128_t::operator+=(const uint128_t &a) {
    uint128_t res;
    res.lo = this->lo + a.lo;
    res.hi = this->hi + a.hi + (res.lo < a.lo);
    this->lo = res.lo;
    this->hi = res.hi;
    return *this;
}

__inline__ __device__ uint128_t &uint128_t::operator-=(const uint128_t &a) {
    uint128_t res;
    res.lo = this->lo - a.lo;
    res.hi = this->hi - a.hi - (res.lo > this->lo);
    this->lo = res.lo;
    this->hi = res.hi;
    return *this;
}

// k = ceil(logq) is at most 61
__device__ uint64_t modBarrett(const uint128_t a, const uint64_t q,
                               const uint64_t mu, const uint64_t twok) {
    const uint64_t k = twok / 2;

    // x1 = t >> (k - 2)
    // t is 2K bits, so x1 is K + 2 bits
    const uint64_t x1 = (a.lo >> (k - 2)) | (a.hi << (64 - k + 2));

    // x2 = x1 * mu
    // mu is K bits, so x2 is 2K + 2 bits
    uint128_t x2;
    x2.hi = __umul64hi(x1, mu);
    x2.lo = x1 * mu;

    // s = x2 >> (k + 2)
    // s is K bits
    const uint64_t s = (x2.lo >> (k + 2)) | (x2.hi << (64 - k - 2));

    // r = s * q
    // r is 2K bits
    uint128_t r;
    r.hi = __umul64hi(s, q);
    r.lo = s * q;

    // c = t - r
    uint64_t c_lo = a.lo - r.lo;

    // if c >= q, c = c - q
    if (c_lo >= q) {
        c_lo -= q;
    }

    return c_lo;
}

__device__ uint64_t modmul(uint64_t a, uint64_t b, uint64_t q, uint64_t mr,
                           uint64_t twok) {
    // a * b
    uint128_t ab;
    ab.hi = __umul64hi(a, b);
    ab.lo = a * b;

    return modBarrett(ab, q, mr, twok);
}

// ===================================
// Phase1
// sPoly = 1 * params->n1 elements
// sPolyBatch = 1 * nyBatchPhase1 sPoly

// Phase2
// sPoly = 1 * params->n2
// sPolyBatch = 1 * nyBatchPhase2 sPoly
// ===================================

__device__ void Add_Phase1(Params *params, uint64_t *dst, const uint64_t *a,
                           const uint64_t *b, const int nyBatch,
                           const bool if_dst_gmem, const bool if_a_gmem,
                           const bool if_b_gmem) {
    for (int i = threadIdx.x; i < nyBatch * params->n1; i += blockDim.x) {
        const int y_idx = i / params->n1;
        const int x_idx = i % params->n1;

        const uint64_t mod = params->qVec[y_idx];
        int dst_idx, a_idx, b_idx;
        if (if_dst_gmem) {
            dst_idx = y_idx * params->N + x_idx * params->n2 + blockIdx.x;
        } else {
            dst_idx = y_idx * params->n1 + x_idx;
        }
        if (if_a_gmem) {
            a_idx = y_idx * params->N + x_idx * params->n2 + blockIdx.x;
        } else {
            a_idx = y_idx * params->n1 + x_idx;
        }
        if (if_b_gmem) {
            b_idx = y_idx * params->N + x_idx * params->n2 + blockIdx.x;
        } else {
            b_idx = y_idx * params->n1 + x_idx;
        }
        uint64_t res = a[a_idx] + b[b_idx];
        if (res >= mod) {
            res -= mod;
        }
        dst[dst_idx] = res;
    }
}

__device__ void Add_Phase2(Params *params, uint64_t *dst, const uint64_t *a,
                           const uint64_t *b, const int nyBatch,
                           const bool if_dst_gmem, const bool if_a_gmem,
                           const bool if_b_gmem) {
    for (int i = threadIdx.x; i < nyBatch * params->n2; i += blockDim.x) {
        const int y_idx = i / params->n2;
        const int x_idx = i % params->n2;

        const uint64_t mod = params->qVec[y_idx];
        int dst_idx, a_idx, b_idx;
        if (if_dst_gmem) {
            dst_idx = y_idx * params->N + blockIdx.x * params->n2 + x_idx;
        } else {
            dst_idx = y_idx * params->n2 + x_idx;
        }
        if (if_a_gmem) {
            a_idx = y_idx * params->N + blockIdx.x * params->n2 + x_idx;
        } else {
            a_idx = y_idx * params->n2 + x_idx;
        }
        if (if_b_gmem) {
            b_idx = y_idx * params->N + blockIdx.x * params->n2 + x_idx;
        } else {
            b_idx = y_idx * params->n2 + x_idx;
        }
        uint64_t res = a[a_idx] + b[b_idx];
        if (res >= mod) {
            res -= mod;
        }
        dst[dst_idx] = res;
    }
}

__device__ void Add_Phase0(Params *params, uint64_t *dst, const uint64_t *a,
                           const uint64_t *b) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < params->N * params->limb; i += gridDim.x * blockDim.x) {
        const int l_idx = i / params->N;
        const int n_idx = i % params->N;
        const uint64_t qi = params->qVec[l_idx];
        const int dst_idx = l_idx * params->N + n_idx;
        const int a_idx = l_idx * params->N + n_idx;
        const int b_idx = l_idx * params->N + n_idx;
        uint64_t res = a[a_idx] + b[b_idx];
        if (res >= qi) {
            res -= qi;
        }
        dst[dst_idx] = res;
    }
}

__device__ void Mult(Params *params, const int n, const int l, uint64_t *dst,
                     const uint64_t *a, const uint64_t *b, const int n_dst,
                     const int n_a, const int n_b) {
    for (int i = threadIdx.x; i < n * l; i += blockDim.x) {
        const int l_idx = i / n;
        const int n_idx = i % n;
        const uint64_t qi = params->qVec[l_idx];
        const uint64_t mu = params->qrVec[l_idx];
        const uint64_t twok = params->qTwok[l_idx];
        const int dst_idx = l_idx * n_dst + n_idx;
        const int a_idx = l_idx * n_a + n_idx;
        const int b_idx = l_idx * n_b + n_idx;
        dst[dst_idx] = modmul(a[a_idx], b[b_idx], qi, mu, twok);
    }
}
__device__ void Add_Elem(Params *params, uint64_t *dst, const uint64_t *a,
                         const uint64_t *b, const int dst_global,
                         const int a_global, const int b_global,
                         const int sPoly_x, const int l_idx, const int n_idx) {
    const uint64_t qi = params->qVec[l_idx];
    // const uint64_t mu = params->qrVec[l_idx];
    // const uint64_t twok = params->qTwok[l_idx];
    const int dst_idx = dst_global * (l_idx * params->N + n_idx) +
                        (1 - dst_global) * threadIdx.x;
    const int a_idx =
        a_global * (l_idx * params->N + n_idx) + (1 - a_global) * threadIdx.x;
    const int b_idx =
        b_global * (l_idx * params->N + n_idx) + (1 - b_global) * threadIdx.x;
    uint64_t res = a[a_idx] + b[b_idx];
    if (res >= qi) {
        res -= qi;
    }
    dst[dst_idx] = res;
}